#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>

#define TIMING
#define N 5000

#ifdef TIMING
double avgCPUTime, avgGPUTime;
double cpuStartTime, cpuEndTime;
#endif 

__global__ void monteCarlo(unsigned  int n, unsigned int* inCirc_d, hiprandState_t* states, unsigned int seed)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (index < n) {
		
		double x, y;

		hiprand_init(seed, index, 0, &states[index]);
		//printf("%d\n", index);
		x = hiprand_uniform_double(&states[index]);
		y = hiprand_uniform_double(&states[index]);

		//printf("%.4f %.4f\n", x, y);

		if ((x*x + y*y) <= 1.0) inCirc_d[index] = 1;
		else inCirc_d = 0;

	}
}

//Used to check if there are any errors launching the kernel
void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main()
{
	/*
	creates cuda rand states and mallocs space for them, each state is thread safe
	*/

	#ifdef TIMING
		avgCPUTime = 0;
		avgGPUTime = 0;
	#endif // TIMING
	
	int blockSize = 512;
	int numBlocks = (N + blockSize - 1) / blockSize;

	hiprandState_t* states;
	hipMalloc((void**)&states, blockSize*numBlocks * sizeof(hiprandState_t));
	
	unsigned int total = 0;
	unsigned int* inCirc_d;
	unsigned int* inCirc = (unsigned int*)malloc(blockSize*numBlocks * sizeof(unsigned int));
	
	hipMalloc((void**)&inCirc_d, blockSize*numBlocks*sizeof(int));
	hipMemset(inCirc_d, 0, blockSize*numBlocks * sizeof(int));
	//CUDAErrorCheck();
	double cpu_estimate;
	
	monteCarlo <<<numBlocks, blockSize >>> (N, inCirc_d, states, time(0));
	
	hipDeviceSynchronize();
	//CUDAErrorCheck();

	hipMemcpy(inCirc, inCirc_d, blockSize*numBlocks * sizeof(int), hipMemcpyDeviceToHost);
	//CUDAErrorCheck();

	for (unsigned int i = 0; i < N; i++) {
		total += inCirc[i];
	}

	double pi_estimate = 4 * total / (double)N;
	
	printf("Total in Circle = %d\nEstimate of Pi = %.4f\n", total,pi_estimate);

	hipFree(states);
	hipFree(inCirc_d);


    return 0;
}