//Taken from https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/

#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>

// CUDA Kernal function to add the elements of two arrays on the GPU
__global__
void add(int n, double *x, double *y)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i+=stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	hipEvent_t start, stop;
	int N = 1 << 24; // 1M elements
	hipEventCreate(&start);
	hipEventCreate(&stop);
	double *x; //= new float[N];
	double *y; //= new float[N];
	hipEventRecord(start);
	hipMallocManaged(&x, N * sizeof(double));
	hipMallocManaged(&y, N * sizeof(double));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	int threadBlockSize = 128;
	int numThreadBlocks = (N+threadBlockSize-1)/threadBlockSize;
	// Run kernel on 1M elements on the CPU
	add<<<numThreadBlocks, threadBlockSize >>>(N, x, y);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	// Check for errors (all values should be 3.0f)
	double maxError = 0.0f;
	for (int i = 0; i < N; i++)
	maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "max error: " << maxError << "\nTime used (ms): " << elapsedTime << std::endl;

	// Free memory
	//delete[] x;
	//delete[] y;
	hipFree(x);
	hipFree(y);

	return 0;
}
