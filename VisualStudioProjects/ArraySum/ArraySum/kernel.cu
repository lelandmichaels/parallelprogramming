#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t sumArray(int *arr, int size, int *out, int threadsPerBlock);

__global__ void sumKernel(int *arr, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < size) {
		arr[i] += arr[i+size];
	}
}

int main()
{
	const int arraySize = 20;
	int a[arraySize];// = { 1, 2, 3, 4, 5,15 };
	int sum = 0;
	for (int i = 0; i < arraySize; i++) {
		a[i] = i;
		sum += a[i];
	}
	printf("Actual sum: %d\n", sum);
	// Add vectors in parallel.
	hipError_t cudaStatus = sumArray(a, arraySize, &sum, 128);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA Sum Array failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


hipError_t sumArray(int *arr, int size, int *out, int threadsPerBlock) {

	printf("Running array sum with %d threads per block.\n", threadsPerBlock);
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		int adjustedSize = size;
		if (adjustedSize % 32 != 0) {
			adjustedSize += 32 - (size % 32);
		}
		//determine number of thread blocks required
		int numThreadBlocks;
		//create array for sum reduction on gpu
		int *gpuArray;
		hipMalloc((void **)&gpuArray, adjustedSize * sizeof(int));
		//initialize array to 0 on gpu
		hipMemset((void*)gpuArray, 0, adjustedSize);
		//copy input data to gpu
		hipMemcpy(gpuArray, arr, size, hipMemcpyHostToDevice);
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
			adjustedSize /= 2;
			numThreadBlocks = (adjustedSize + threadsPerBlock - 1) / threadsPerBlock;
			printf("Adjusted size:%d\tThreadBlocks:%d\n", adjustedSize, numThreadBlocks);
			// Launch a kernel on the GPU with one thread for each pair of elements.
			sumKernel << <numThreadBlocks, threadsPerBlock >> > (gpuArray, adjustedSize);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
			else {
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					break;
				}
				hipMemcpy(out, gpuArray, sizeof(int), hipMemcpyDeviceToHost);
				printf("PartialResult:%d\n", *out);
			}
		}
		hipFree(gpuArray);
	}
	return cudaStatus;
}