#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define TIMING

hipError_t sumArray(long long *arr, int size, long long *out, int threadsPerBlock);

__global__ void sumKernel(long long *arr, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < size) {
		arr[i] += arr[i + size];
		arr[i + size] = 0;
	}
}

int main()
{
	int arraySize = 1000000;
	long long *a = (long long*)malloc(arraySize * sizeof(long long));
	long long sum = 0;
#ifdef TIMING
	double startTime = omp_get_wtime(), endTime;
#endif // TIMING
	for (int i = 0; i < arraySize; i++) {
		a[i] = rand();
		sum += a[i];
	}
#ifdef TIMING
	endTime = omp_get_wtime();
	double timeUsed = endTime - startTime;
#endif // TIMING
	printf("Actual sum: %lld.\n", sum);
#ifdef TIMING
	printf("CPU Calculated in %lf ms.\n", timeUsed*1000);
#endif // TIMING
	// Add vectors in parallel.
	hipError_t cudaStatus = sumArray(a, arraySize, &sum, 256);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA Sum Array failed!");
		return 1;
	}
	printf("Cuda sum: %lld.\n", sum);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


hipError_t sumArray(long long *arr, int size, long long *out, int threadsPerBlock) {
#ifdef TIMING
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		//determine number of thread blocks required
		int numThreadBlocks;
		//create array for sum reduction on gpu
		long long *gpuArray;
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		hipMalloc((void **)&gpuArray, adjustedSize * sizeof(long long));
		cudaStatus = hipDeviceSynchronize();
		hipMemset((void*)gpuArray, 0, adjustedSize * sizeof(long long));
		cudaStatus = hipDeviceSynchronize();
		//copy input data to gpu
		hipMemcpy(gpuArray, arr, size * sizeof(long long), hipMemcpyHostToDevice);
		cudaStatus = hipDeviceSynchronize();
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
#ifdef TIMING
			hipEventRecord(start);
			hipEventSynchronize(start);
#endif // TIMING
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize > 1) {
				adjustedSize++;
			}
			if (adjustedSize < threadsPerBlock) {
				threadsPerBlock = adjustedSize;
				if (threadsPerBlock % 32 != 0) {
					threadsPerBlock += 32 - threadsPerBlock % 32;
				}
			}
			numThreadBlocks = (adjustedSize + threadsPerBlock - 1) / threadsPerBlock;
			//printf("Adjusted size:%d\tThreadBlocks:%d\tThreadsPerBlock:%d\n", adjustedSize, numThreadBlocks,threadsPerBlock);
			// Launch a kernel on the GPU with one thread for each pair of elements.
			sumKernel << <numThreadBlocks, threadsPerBlock >> > (gpuArray, adjustedSize);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
			else {
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					break;
				}
				hipMemcpy(out, gpuArray, sizeof(long long), hipMemcpyDeviceToHost);
			}
#ifdef TIMING
			hipEventRecord(end);
			hipEventSynchronize(end);
			hipEventElapsedTime(&gpuTimeUsed, start, end);
			totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
		}
#ifdef TIMING
		printf("GPU Calculated in %lf ms.\n", totalGpuTimeUsed);
#endif // TIMING

		hipFree(gpuArray);
	}
	return cudaStatus;
}