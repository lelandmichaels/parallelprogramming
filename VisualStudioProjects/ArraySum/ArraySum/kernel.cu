#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

hipError_t sumArray(long long *arr, int size, long long *out, int threadsPerBlock);

__global__ void sumKernel(long long *arr, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < size) {
		//printf("Adding arr[%d]=%d and arr[%d]=%d\n", i, arr[i], i + size, arr[i + size]);
		arr[i] += arr[i + size];
		arr[i + size] = 0;
	}
}

int main()
{
	const int arraySize = 3000;
	long long a[arraySize];// = { 1, 2, 3, 4, 5,15 };
	long long sum = 0;
	for (int i = 0; i < arraySize; i++) {
		a[i] = rand();
		sum += a[i];
		//printf("i:%d\tsum so far:%d\n", i, sum);
	}
	printf("Actual sum: %lld\n", sum);
	// Add vectors in parallel.
	hipError_t cudaStatus = sumArray(a, arraySize, &sum, 128);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA Sum Array failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


hipError_t sumArray(long long *arr, int size, long long *out, int threadsPerBlock) {

	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		//determine number of thread blocks required
		int numThreadBlocks;
		//create array for sum reduction on gpu
		long long *gpuArray;
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		hipMalloc((void **)&gpuArray, adjustedSize * sizeof(long long));
		cudaStatus = hipDeviceSynchronize();
		hipMemset((void*)gpuArray, 0, adjustedSize * sizeof(long long));
		cudaStatus = hipDeviceSynchronize();
		//copy input data to gpu
		hipMemcpy(gpuArray, arr, size * sizeof(long long), hipMemcpyHostToDevice);
		cudaStatus = hipDeviceSynchronize();
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize>1) {
				adjustedSize++;
			}
			if (adjustedSize < threadsPerBlock) {
				threadsPerBlock = adjustedSize;
				if (threadsPerBlock % 32 != 0) {
					threadsPerBlock += 32 - threadsPerBlock % 32;
				}
			}
			numThreadBlocks = (adjustedSize + threadsPerBlock - 1) / threadsPerBlock;
			//printf("Adjusted size:%d\tThreadBlocks:%d\tThreadsPerBlock:%d\n", adjustedSize, numThreadBlocks,threadsPerBlock);
			// Launch a kernel on the GPU with one thread for each pair of elements.
			sumKernel << <numThreadBlocks, threadsPerBlock >> > (gpuArray, adjustedSize);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
			else {
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					break;
				}
				hipMemcpy(out, gpuArray, sizeof(long long), hipMemcpyDeviceToHost);
			}
		}
		printf("PartialResult:%lld\n", *out);
		hipFree(gpuArray);
	}
	return cudaStatus;
}