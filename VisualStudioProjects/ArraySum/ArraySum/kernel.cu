#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t sumArray(int *arr, long long *out, unsigned long long size, int threadsPerBlock, int numBlocks);

__global__ void sumKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int arraySize = 5;
	int a[arraySize] = { 1, 2, 3, 4, 5 };
	long long sum = 0;
	// Add vectors in parallel.
	hipError_t cudaStatus = sumArray(a, &sum, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA Sum Array failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


hipError_t sumArray(int *arr, long long *out, unsigned long long size, int threadsPerBlock, int numBlocks) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		// Launch a kernel on the GPU with one thread for each element.
		sumKernel<<<1, size>>>(dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		else {
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			}
		}
	}
	return cudaStatus;
}