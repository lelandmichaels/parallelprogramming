#include "hip/hip_runtime.h"
#include "GPU_Sum.h"

#define NUM_THREADS 8
#define TIMING
#define MIN_SIZE 12500
#define SIZE_INCREMENT 12500
#define MAX_SIZE 1000000
#define SAMPLE_SIZE 50

#ifdef TIMING
double avgCPUTime, avgGPUTime;
double cpuStartTime, cpuEndTime;
#endif // TIMING

template<typename Type>
hipError_t sumArray(Type *arr, int size, Type *out, int blockCount, int threadsPerBlock, bool arrayOnGPU);

template<typename Type>
__global__ void sumKernel(Type *arr, int size)
{
	int stride = blockDim.x * gridDim.x;
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	long long temp = arr[id];
	for (int i = id + stride; i < size; i += stride) {
		temp += arr[i];
	}
	arr[id] = temp;
}

template<typename Type>
__global__ void binaryReduction(Type *arr, int size)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if (id < size) {
		arr[id] += arr[id + size];
	}
}

int main()
{
	long long *a = (long long*)malloc(MAX_SIZE * sizeof(long long));
	for (int i = 0; i < MAX_SIZE; i++) {
		a[i] = i;// rand();
	}
	int timesCorrect = 0, timesWrong = 0;
	printf("Size\tAvgCPUTime\tAvgGPUTime\tSamples:%d\n", SAMPLE_SIZE);
	for (int arraySize = MIN_SIZE; arraySize <= MAX_SIZE; arraySize += SIZE_INCREMENT) {
#ifdef TIMING
		avgCPUTime = 0;
		avgGPUTime = 0;
#endif // TIMING
		long long sum = 0, cudaSum;
#pragma omp parallel for num_threads(NUM_THREADS) \
reduction(+:avgGPUTime,avgCPUTime,timesCorrect,timesWrong) \
private(cpuStartTime,cpuEndTime)
		for (int sample = 0; sample < SAMPLE_SIZE; sample++) {
			sum = 0;
#ifdef TIMING
			cpuStartTime = omp_get_wtime();
#endif // TIMING
			for (int i = 0; i < arraySize; i++) {
				sum += a[i];
			}
#ifdef TIMING
			cpuEndTime = omp_get_wtime();
			double timeUsed = 1000 * (cpuEndTime - cpuStartTime);
			avgCPUTime += timeUsed;
#endif // TIMING
			// Add vectors in parallel.
			hipError_t cudaStatus = sumArray(a, arraySize, &cudaSum, 1, 256, false);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "CUDA Sum Array failed!");
				return 1;
			}

			if (cudaSum == sum) {
				timesCorrect++;
			}
			else {
				timesWrong++;
			}

			// hipDeviceReset must be called before exiting in order for profiling and
			// tracing tools such as Nsight and Visual Profiler to show complete traces.
			cudaStatus = hipDeviceReset();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceReset failed!");
				return 1;
			}
		}
#ifdef TIMING
		printf("%d\t%lf\t%lf\n", arraySize, avgCPUTime / SAMPLE_SIZE, avgGPUTime / SAMPLE_SIZE);
#endif // TIMING
	}
	printf("GPU Implementation was correct %d times and incorrect %d times.\n", timesCorrect, timesWrong);
	return 0;
}


template<typename Type>
hipError_t sumArray(Type *arr, int size, Type *out, int blockCount, int blockSize, bool arrayOnGPU) {
#ifdef TIMING
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
	hipEvent_t startStep, endStep;
	hipEventCreateWithFlags(&startStep, hipEventDefault);
	hipEventCreateWithFlags(&endStep, hipEventDefault);
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		hipStream_t myStream;
		hipStreamCreateWithFlags(&myStream, hipStreamNonBlocking);
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		//determine number of thread blocks required
		//create array for sum reduction on gpu
		Type *gpuArray;
		if (arrayOnGPU) {
			gpuArray = arr;
		}
		else {
			hipMalloc((void **)&gpuArray, adjustedSize * sizeof(Type));
			cudaStatus = hipStreamSynchronize(myStream);
			hipMemsetAsync((void*)gpuArray, 0, adjustedSize * sizeof(Type), myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			//copy input data to gpu
			hipMemcpyAsync(gpuArray, arr, size * sizeof(Type), hipMemcpyHostToDevice, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
		}
		sumKernel << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);
		adjustedSize = blockCount*blockSize;
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
			cudaStatus = hipStreamSynchronize(myStream);
#ifdef TIMING
			hipEventRecord(startStep, myStream);
#endif // TIMING
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize > 1) {
				adjustedSize++;
			}
			// Launch a kernel on the GPU with one thread for each pair of elements.
			binaryReduction << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
#ifdef TIMING
			hipEventRecord(endStep, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
			totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
		}
#ifdef TIMING
		avgGPUTime += totalGpuTimeUsed;
#endif // TIMING
		if (cudaStatus == hipSuccess) {
			hipMemcpyAsync(out, gpuArray, sizeof(Type), hipMemcpyDeviceToHost, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			}
		}
		hipStreamDestroy(myStream);
		if (arrayOnGPU) {
			hipFree(gpuArray);
		}
	}
	return cudaStatus;
}