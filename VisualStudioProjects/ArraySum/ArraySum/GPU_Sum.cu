#include "hip/hip_runtime.h"
#include "GPU_Sum.h"

#define NUM_THREADS 1
#define TIMING
#define MIN_SIZE 12500
#define SIZE_INCREMENT 2
#define MAX_SIZE 10000000
#define SAMPLE_SIZE 50

#define BLOCKS_MIN 1
#define BLOCKS_INCREMENT 2
#define BLOCKS_MAX 128


#ifdef TIMING
double avgCPUTimeMS, avgGPUTimeMS;
double cpuStartTime, cpuEndTime;
#endif // TIMING

template<typename Type>
__global__ void sumKernel(Type *arr, int size)
{
	int stride = blockDim.x * gridDim.x;
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	long long temp = arr[id];
	for (int i = id + stride; i < size; i += stride) {
		temp += arr[i];
	}
	arr[id] = temp;
}

template<typename Type>
__global__ void binaryReduction(Type *arr, int size)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if (id < size) {
		arr[id] += arr[id + size];
	}
}

int main()
{
	long long *a = (long long*)malloc(MAX_SIZE * sizeof(long long));
#pragma omp parallel for num_threads(8)
	for (int i = 0; i < MAX_SIZE; i++) {
		a[i] = i;// rand();
	}
	printf("Size\tCPU");
	for (int blocks = BLOCKS_MIN; blocks <= BLOCKS_MAX; blocks *= BLOCKS_INCREMENT) {
		printf("\t%d", blocks);
	}
	printf("\n");
	for (int arraySize = MIN_SIZE; arraySize <= MAX_SIZE; arraySize *= SIZE_INCREMENT) {
		printf("%ld", arraySize);
#ifdef TIMING
		avgCPUTimeMS = 0;
		cpuStartTime = omp_get_wtime();
#endif // TIMING
		for (int sample = 0; sample < SAMPLE_SIZE; sample++) {
			long long sum = 0;
#pragma omp parallel num_threads(1)
			for (int i = 0; i < arraySize; i++) {
				sum += a[i];
			}
		}
#ifdef TIMING
		cpuEndTime = omp_get_wtime();
		double timeUsedS = (cpuEndTime - cpuStartTime);
		avgCPUTimeMS += timeUsedS * 1000.0;
		printf("\t%lf", avgCPUTimeMS);// / SAMPLE_SIZE);
#endif // TIMING
		for (int blocks = BLOCKS_MIN; blocks <= BLOCKS_MAX; blocks *= BLOCKS_INCREMENT) {
#ifdef TIMING
			avgGPUTimeMS = 0;
#endif // TIMING
			long long cudaSum;
			for (int sample = 0; sample < SAMPLE_SIZE; sample++) {
				hipError_t cudaStatus = sumArray(a, arraySize, &cudaSum, blocks, 32, false, true);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "CUDA Sum Array failed!");
					return 1;
				}
				// hipDeviceReset must be called before exiting in order for profiling and
				// tracing tools such as Nsight and Visual Profiler to show complete traces.
				cudaStatus = hipDeviceReset();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
			}
#ifdef TIMING
			printf("\t%lf", avgGPUTimeMS / SAMPLE_SIZE);
			avgGPUTimeMS = 0;
#endif // TIMING
		}
		printf("\n");
	}
	//printf("GPU Implementation was correct %d times and incorrect %d times.\n", timesCorrect, timesWrong);
	return 0;
}


template<typename Type>
hipError_t sumArray(Type *arr, int size, Type *out, int blockCount, int blockSize, bool arrayOnGPU, bool createNewStream) {
	hipStream_t myStream = (hipStream_t)(0);
	if (createNewStream) {
		hipStreamCreateWithFlags(&myStream, hipStreamDefault);
	}
#ifdef TIMING
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
	hipEvent_t startStep, endStep;
	hipEventCreateWithFlags(&startStep, hipEventDefault);
	hipEventCreateWithFlags(&endStep, hipEventDefault);
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		//determine number of thread blocks required
		//create array for sum reduction on gpu
		Type *gpuArray;
		if (arrayOnGPU) {
			gpuArray = arr;
		}
		else {
			hipMalloc((void **)&gpuArray, adjustedSize * sizeof(Type));
			cudaStatus = hipStreamSynchronize(myStream);
			hipMemsetAsync((void*)gpuArray, 0, adjustedSize * sizeof(Type), myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			//copy input data to gpu
			hipMemcpyAsync(gpuArray, arr, size * sizeof(Type), hipMemcpyHostToDevice, myStream);
			//cudaStatus = hipStreamSynchronize(myStream);
		}
#ifdef TIMING
		hipEventRecord(startStep, myStream);
#endif // TIMING
		sumKernel << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);
#ifdef TIMING
		hipEventRecord(endStep, myStream);
		cudaStatus = hipStreamSynchronize(myStream);
		hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
		totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
		adjustedSize = blockCount*blockSize;
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
			cudaStatus = hipStreamSynchronize(myStream);
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize > 1) {
				adjustedSize++;
			}
#ifdef TIMING
			hipEventRecord(startStep, myStream);
			cudaStatus = hipEventSynchronize(startStep);
#endif // TIMING
			// Launch a kernel on the GPU with one thread for each pair of elements.
			binaryReduction << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);
#ifdef TIMING
			hipEventRecord(endStep, myStream);
			cudaStatus = hipEventSynchronize(endStep);
			hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
			totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
		}
#ifdef TIMING
		avgGPUTimeMS += totalGpuTimeUsed;
#endif // TIMING
		if (cudaStatus == hipSuccess) {
			hipMemcpyAsync(out, gpuArray, sizeof(Type), hipMemcpyDeviceToHost, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			}
		}
		if (arrayOnGPU) {
			hipFree(gpuArray);
		}
	}
	if (createNewStream) {
		hipStreamDestroy(myStream);
	}
	return cudaStatus;
}