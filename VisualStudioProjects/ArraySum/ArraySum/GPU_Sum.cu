#include "hip/hip_runtime.h"
#include "GPU_Sum.h"

#define NUM_THREADS 1
#define TIMING
#define MIN_SIZE 125000
#define SIZE_INCREMENT 2
#define MAX_SIZE 100000000
#define SAMPLE_SIZE 50

#define BLOCKS_MIN 1
#define BLOCKS_INCREMENT 1
#define BLOCKS_MAX 256


#ifdef TIMING
double avgCPUTime, avgGPUTime;
double cpuStartTime, cpuEndTime;
#endif // TIMING

template<typename Type>
__global__ void sumKernel(Type *arr, int size)
{
	int stride = blockDim.x * gridDim.x;
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	long long temp = arr[id];
	for (int i = id + stride; i < size; i += stride) {
		temp += arr[i];
	}
	arr[id] = temp;
}

template<typename Type>
__global__ void binaryReduction(Type *arr, int size)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if (id < size) {
		arr[id] += arr[id + size];
	}
}

int main()
{
	long long *a = (long long*)malloc(MAX_SIZE * sizeof(long long));
	for (int i = 0; i < MAX_SIZE; i++) {
		a[i] = i;// rand();
	}
	int timesCorrect = 0, timesWrong = 0;
	printf("Size");
	for (int blocks = BLOCKS_MIN; blocks < BLOCKS_MAX; blocks += BLOCKS_INCREMENT) {
		printf("\t%d",blocks);
	}
	printf("\n");
	for (int arraySize = MIN_SIZE; arraySize <= MAX_SIZE; arraySize *= SIZE_INCREMENT) {
		printf("%ld", arraySize);
		for (int blocks = BLOCKS_MIN; blocks < BLOCKS_MAX; blocks += BLOCKS_INCREMENT) {
#ifdef TIMING
			avgCPUTime = 0;
			avgGPUTime = 0;
#endif // TIMING
			long long sum = 0, cudaSum;
#pragma omp parallel for num_threads(NUM_THREADS) \
reduction(+:avgGPUTime,avgCPUTime,timesCorrect,timesWrong) \
private(cpuStartTime,cpuEndTime)
			for (int sample = 0; sample < SAMPLE_SIZE; sample++) {
				sum = 0;
#ifdef TIMING
				cpuStartTime = omp_get_wtime();
#endif // TIMING
				/*for (int i = 0; i < arraySize; i++) {
					sum += a[i];
				}*/
#ifdef TIMING
				cpuEndTime = omp_get_wtime();
				double timeUsed = 1000 * (cpuEndTime - cpuStartTime);
				avgCPUTime += timeUsed;
#endif // TIMING
				// Add vectors in parallel.
				hipError_t cudaStatus = sumArray(a, arraySize, &cudaSum, blocks);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "CUDA Sum Array failed!");
					return 1;
				}

				/*if (cudaSum == sum) {
					timesCorrect++;
				}
				else {
					timesWrong++;
				}*/

				// hipDeviceReset must be called before exiting in order for profiling and
				// tracing tools such as Nsight and Visual Profiler to show complete traces.
				cudaStatus = hipDeviceReset();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
			}
#ifdef TIMING
			printf("\t%lf", avgGPUTime / SAMPLE_SIZE);
			avgGPUTime = 0;
		}
#endif // TIMING
		printf("\n");
	}
	//printf("GPU Implementation was correct %d times and incorrect %d times.\n", timesCorrect, timesWrong);
	return 0;
}


template<typename Type>
hipError_t sumArray(Type *arr, int size, Type *out, int blockCount, int blockSize, bool arrayOnGPU, hipStream_t myStream) {
#ifdef TIMING
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
	hipEvent_t startStep, endStep;
	hipEventCreateWithFlags(&startStep, hipEventDefault);
	hipEventCreateWithFlags(&endStep, hipEventDefault);
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		//determine number of thread blocks required
		//create array for sum reduction on gpu
		Type *gpuArray;
		if (arrayOnGPU) {
			gpuArray = arr;
		}
		else {
			hipMalloc((void **)&gpuArray, adjustedSize * sizeof(Type));
			cudaStatus = hipStreamSynchronize(myStream);
			hipMemsetAsync((void*)gpuArray, 0, adjustedSize * sizeof(Type), myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			//copy input data to gpu
			hipMemcpyAsync(gpuArray, arr, size * sizeof(Type), hipMemcpyHostToDevice, myStream);
			//cudaStatus = hipStreamSynchronize(myStream);
		}
#ifdef TIMING
		hipEventRecord(startStep, myStream);
#endif // TIMING
		sumKernel << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);
#ifdef TIMING
		hipEventRecord(endStep, myStream);
		cudaStatus = hipStreamSynchronize(myStream);
		hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
		totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
		adjustedSize = blockCount*blockSize;
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
			cudaStatus = hipStreamSynchronize(myStream);
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize > 1) {
				adjustedSize++;
			}
#ifdef TIMING
			hipEventRecord(startStep, myStream);
			cudaStatus = hipEventSynchronize(startStep);
#endif // TIMING
			// Launch a kernel on the GPU with one thread for each pair of elements.
			binaryReduction << <blockCount, blockSize, 0, myStream >> > (gpuArray, adjustedSize);
#ifdef TIMING
			hipEventRecord(endStep, myStream);
			cudaStatus = hipEventSynchronize(endStep);
			hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
			totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
		}
#ifdef TIMING
		avgGPUTime += totalGpuTimeUsed;
#endif // TIMING
		if (cudaStatus == hipSuccess) {
			hipMemcpyAsync(out, gpuArray, sizeof(Type), hipMemcpyDeviceToHost, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			}
		}
		if (arrayOnGPU) {
			hipFree(gpuArray);
		}
	}
	return cudaStatus;
}