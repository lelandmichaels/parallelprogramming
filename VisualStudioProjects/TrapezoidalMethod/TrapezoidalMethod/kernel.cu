#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <cstdlib>

#define TIMING

#ifdef TIMING
double avgCPUTime, avgGPUTime;
double cpuStartTime, cpuEndTime;
#endif // TIMING


// Kernel function to calculate trapizoidal sum
__host__
__device__
double f(double x) {
	return x*x;
}

__global__
void trap(int a, int n, double h, double* sum) {
	double x_i;
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;
	for (int i = id; i < n; i += stride) {
		x_i = a + i*h;
		sum[id] += f(x_i);
	}
}

hipError_t sumArray(double *arr, int size, double *out, int threadsPerBlock, bool arrayAlreadyOnGPU);

__global__ void sumKernel(double *arr, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < size) {
		arr[i] += arr[i + size];
		arr[i + size] = 0;
	}
}

hipError_t trapezoidalMethod(double start, double end, int subdivisions, double *out, int blockCount, int blockSize) {
#ifdef TIMING
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
	hipEvent_t startStep, endStep;
	hipEventCreateWithFlags(&startStep, hipEventDefault);//hipEventBlockingSync);
	hipEventCreateWithFlags(&endStep, hipEventDefault);// hipEventBlockingSync);
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		hipStream_t myStream;
		hipStreamCreateWithFlags(&myStream, hipStreamNonBlocking);
#ifdef TIMING
		hipEventRecord(startStep, myStream);
		hipStreamSynchronize(myStream);
#endif // TIMING
		// Launch a kernel on the GPU with one thread for each element.
		double  h = (end - start) / (double)subdivisions;
		double *gpuSum, cpuSum;
		hipMalloc(&gpuSum, sizeof(double)*blockCount*blockSize);
		hipStreamSynchronize(myStream);
		trap<<<blockCount, blockSize, 0, myStream >>>(start, subdivisions, h, gpuSum);
		hipStreamSynchronize(myStream);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Trapezoidal launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		else {
			//reduce array to one value now
			sumArray(gpuSum, blockCount*blockSize, &cpuSum, blockSize, true);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "Reduction failed: %s\n", hipGetErrorString(cudaStatus));
			}
			else {
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				hipMemcpyAsync(&cpuSum, gpuSum, sizeof(double), hipMemcpyDeviceToHost, myStream);
				cudaStatus = hipStreamSynchronize(myStream);
				hipFree(gpuSum);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Trapezoidal!\n", cudaStatus);
				}
				else {
					*out = (f(start) + f(end)) / 2.0 + cpuSum;
					*out *= h;
				}
			}
		}
#ifdef TIMING
		hipEventRecord(endStep, myStream);
		hipStreamSynchronize(myStream);
		hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
		totalGpuTimeUsed += gpuTimeUsed;
		avgGPUTime += totalGpuTimeUsed;
#endif // TIMING
		hipStreamDestroy(myStream);
	}
	return cudaStatus;
}

int main(void) {
#ifdef TIMING
	avgCPUTime = 0;
	avgGPUTime = 0;
#endif // TIMING
	int a = 1;
	int b = 2;
	int n = 100;
	double sum = 0.0;
	hipError_t trapezoidalLaunch = trapezoidalMethod(a, b, n, &sum, 5, 256);
	if (trapezoidalLaunch == hipSuccess) {
		printf("%lf\n", sum);
	}
	else {
		printf("There was an error runnning the operation.\n");
		printf("Error code: %d\n", trapezoidalLaunch);
	}
	return 0;
}


hipError_t sumArray(double *arr, int size, double *out, int threadsPerBlock, bool arrayAlreadyOnGPU) {
#ifdef TIMING
	double totalGpuTimeUsed = 0;
	float gpuTimeUsed;
	hipEvent_t startStep, endStep;
	hipEventCreateWithFlags(&startStep, hipEventDefault);
	hipEventCreateWithFlags(&endStep, hipEventDefault);
#endif // TIMING
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		hipStream_t myStream;
		hipStreamCreateWithFlags(&myStream, hipStreamNonBlocking);
		//adjust size so it is even
		int adjustedSize = size + size % 2;
		//determine number of thread blocks required
		int numThreadBlocks = numThreadBlocks = (adjustedSize + threadsPerBlock - 1) / threadsPerBlock;
		if (numThreadBlocks > 65535) {
			double part1, part2;
			sumArray(arr, size / 2, &part1, threadsPerBlock,arrayAlreadyOnGPU);
			sumArray(&arr[size / 2], size / 2, &part2, threadsPerBlock, arrayAlreadyOnGPU);
			cudaStatus = hipGetLastError();
			if (cudaStatus == hipSuccess) {
				*out = part1 + part2;
			}
			return cudaStatus;
		}
		double *gpuArray;
		if (arrayAlreadyOnGPU) {
			gpuArray = arr;
		}
		else {
			//create array for sum reduction on gpu
			hipMalloc((void **)&gpuArray, adjustedSize * sizeof(double));
			cudaStatus = hipStreamSynchronize(myStream);
			hipMemsetAsync((void*)gpuArray, 0, adjustedSize * sizeof(double), myStream);
			cudaStatus = hipStreamSynchronize(myStream);
			//copy input data to gpu
			hipMemcpyAsync(gpuArray, arr, size * sizeof(double), hipMemcpyHostToDevice, myStream);
			cudaStatus = hipStreamSynchronize(myStream);
		}
		//keep reducing the problem size by two
		while (adjustedSize > 1) {
#ifdef TIMING
			hipEventRecord(startStep, myStream);
			hipStreamSynchronize(myStream);
			//hipStreamWaitEvent(myStream, startStep, 0);
#endif // TIMING
			adjustedSize /= 2;
			if (adjustedSize % 2 != 0 && adjustedSize > 1) {
				adjustedSize++;
			}
			if (adjustedSize < threadsPerBlock) {
				threadsPerBlock = adjustedSize;
				if (threadsPerBlock % 32 != 0) {
					threadsPerBlock += 32 - threadsPerBlock % 32;
				}
			}
			numThreadBlocks = (adjustedSize + threadsPerBlock - 1) / threadsPerBlock;
			//printf("Adjusted size:%d\tThreadBlocks:%d\tThreadsPerBlock:%d\n", adjustedSize, numThreadBlocks,threadsPerBlock);
			// Launch a kernel on the GPU with one thread for each pair of elements.
			sumKernel << <numThreadBlocks, threadsPerBlock, 0, myStream >> > (gpuArray, adjustedSize);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "sumArray Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				break;
			}
			else {
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				hipStreamSynchronize(myStream);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					break;
				}
				hipMemcpyAsync(out, gpuArray, sizeof(long long), hipMemcpyDeviceToHost, myStream);
			}
#ifdef TIMING
			hipEventRecord(endStep, myStream);
			hipStreamSynchronize(myStream);
			//hipStreamWaitEvent(myStream,endStep,0);
			hipEventElapsedTime(&gpuTimeUsed, startStep, endStep);
			totalGpuTimeUsed += gpuTimeUsed;
#endif // TIMING
		}
#ifdef TIMING
		avgGPUTime += totalGpuTimeUsed;
#endif // TIMING
		hipStreamDestroy(myStream);
		if (!arrayAlreadyOnGPU) {
			hipFree(gpuArray);
		}
	}
	return cudaStatus;
}