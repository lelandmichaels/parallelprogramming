#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <cstdlib>
// Kernel function to calculate trapizoidal sum
__host__
__device__
double f(double x) {
	return x*x;
}

__global__
void trap(int a, int n, double h, double* sum) {
	double x_i;
	for (int i = 1; i < n; i++) {
		x_i = a + i*h;
		*sum += f(x_i);
	}
}

hipError_t trapezoidalMethod(double start, double end, int subdivisions, double *out) {
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else {
		// Launch a kernel on the GPU with one thread for each element.
		double  h = (end - start) / (double)subdivisions;
		double sum = 0.0;
		sum += (f(start) + f(end)) / 2.0;
		trap << <1, 1 >> > (start, subdivisions, h, &sum);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Trapezoidal launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		else {
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			sum *= h;
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Trapezoidal!\n", cudaStatus);
			}
		}
	}
	return cudaStatus;
}

int main(void) {

	int a = 1;
	int b = 2;
	int n = 100;
	double sum = 0.0;
	hipError_t trapezoidalLaunch = trapezoidalMethod(a, b, n, &sum);
	if (trapezoidalLaunch == hipSuccess) {
		printf("%lf\n", sum);
	}
	else {
		printf("There was an error runnning the operation.\n");
		printf("Error code: %d\n", trapezoidalLaunch);
	}
	return 0;
}
