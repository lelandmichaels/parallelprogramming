#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <omp.h>
#include <cstdlib>

#define TIMING
#define MIN_SIZE 250000
#define SIZE_INCREMENT 250000
#define MAX_SIZE 10000000
#define SAMPLE_SIZE 1

#ifdef TIMING
double avgCPUTime, avgGPUTime;
double cpuStartTime, cpuEndTime;
#endif // TIMING

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif
// Kernel function to calculate trapizoidal sum
__host__
__device__
double f(int x, double* arr1, double* arr0) {
  return double ans = (.01*(arr1[x-1] - (2*arr1[x]) + arr1[x+1]) + 2* arr1[x] - arr0[x]);
}

__global__
void wave(int n, double *arr0, double* arr1, double* arr2) {
  for(int i = 0; i < n; i++){
    arr2[i] = f(i, arr1, arr0);
  }
}

int main(void) {


  	hipEvent_t start, stop;
	int N = 100; // 1M elements
	int steps = 10;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	double *arr0; //= new float[N];
	double *arr1; //= new float[N];
	double *arr2; //= new float[N];
	double *temp;
	hipEventRecord(start);
	hipMallocManaged(&arr0, N * sizeof(double));
	hipMallocManaged(&arr1, N * sizeof(double));
	hipMallocManaged(&arr2, N * sizeof(double));

	arr1[0] = 0;
	arr1[N-1] = 0;
	arr0[0] = 0;
	arr0[N-1] = 0;

	// initialize arr1 and arr2 arrays
	for (int i = 1; i < N-2; i++) {
	  arr1[i] = sin(M_PI*i);
	  arr0[i] = sin(M_PI*i);
	}
	
	int threadBlockSize = 128;
	int numThreadBlocks = (N+threadBlockSize-1)/threadBlockSize;
	// Run kernel on 1M elements on the CPU
	for(int i = 0; i < steps; i++){
	  wave<<<numThreadBlocks, threadBlockSize >>>(N, arr0, arr1, arr2);
	  
	  temp = arr0;
	  arr0 = arr1;
	  arr1 = arr2;
	  arr2 = temp;
	}
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	// Free memory
	//delete[] x;
	//delete[] y;
	hipFree(arr0);
	hipFree(arr1);
	hipFree(arr2);

	return 0;
}
