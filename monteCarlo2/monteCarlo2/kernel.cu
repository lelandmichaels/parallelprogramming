#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "GPU_Sum.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>

#define TIMING

/*__global__ void monteCarlo(unsigned  int n, unsigned int* inCirc_d, hiprandState_t* states, unsigned int seed)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {

		double x, y;

		hiprand_init(seed, index, 0, &states[index]);
		//printf("%d\n", index);
		x = hiprand_uniform_double(&states[index]);
		y = hiprand_uniform_double(&states[index]);

		//printf("%.4f %.4f\n", x, y);

		if ((x*x + y*y) <= 1.0) inCirc_d[index] = 1;
		else inCirc_d = 0;

	}
}*/

__global__ void monteCarlo(unsigned  int n, unsigned int* inCirc_d, hiprandState_t* states, unsigned int seed)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x*gridDim.x;
	

	for(unsigned int i = index; i < n; i += stride) {

		double x, y;

		hiprand_init(seed, i, 0, &states[i]);
		//printf("%d\n", index);
		x = hiprand_uniform_double(&states[i]);
		y = hiprand_uniform_double(&states[i]);

		//printf("%.4f %.4f\n", x, y);

		if ((x*x + y*y) <= 1.0) inCirc_d[index] = 1;
		else inCirc_d = 0;

	}
}

//Used to check if there are any errors launching the kernel
void CUDAErrorCheck()
{	
	
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main()
{
	/*
	creates cuda rand states and mallocs space for them, each state is thread safe
	*/
	
	int N = 1000000;
	int blockSize = 512;
	int numBlocks = (N + blockSize - 1) / blockSize;
	bool onGpu = false;

	hiprandState_t* states;
	hipMalloc((void**)&states, N * sizeof(hiprandState_t));

	unsigned int total = 0;
	unsigned int* inCirc_d;
	unsigned int* inCirc = (unsigned int*)malloc(N * sizeof(unsigned int));


	hipMalloc((void**)&inCirc_d, N * sizeof(unsigned int));
	hipMemset(inCirc_d, 0, N * sizeof(unsigned int));
	//CUDAErrorCheck();
	double cpu_estimate;

	monteCarlo << <numBlocks, blockSize >> > (N, inCirc_d, states, time(0));

	hipDeviceSynchronize();
	//CUDAErrorCheck();

	hipMemcpy(inCirc, inCirc_d, N * sizeof(int), hipMemcpyDeviceToHost);
	//CUDAErrorCheck();
	hipDeviceSynchronize();

	//sumArray(inCirc, N, &total, numBlocks, blockSize, onGpu);
	
	for (unsigned int i = 0; i < N; i++) {
		total += inCirc[i];
		//printf("i = %d and count = %d\n",i,inCirc[i]);
	}


	double pi_estimate = 4 * total / (double)N;

	printf("Total in Circle = %d\nEstimate of Pi = %.4f\n", total, pi_estimate);

	hipFree(states);
	hipFree(inCirc_d);


	return 0;
}

