#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "GPU_Sum.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <omp.h>
#include <string.h>

#define TIMING


__global__ void monteCarlo(long long n, int* inCirc_d, hiprandState_t* states, unsigned int seed)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;
	double x, y;
	long long myTotal = 0;
	hiprand_init(seed+index, 0/*index*/, 0, &states[index]);
	for (long long i = index; i < n; i += stride) {
		x = hiprand_uniform_double(&states[index]);
		y = hiprand_uniform_double(&states[index]);
		myTotal += ((x*x + y*y) <= 1.0);
	}
	inCirc_d[index] += myTotal;
}

//Used to check if there are any errors launching the kernel
void CUDAErrorCheck()
{

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main()
{
	/*
	creates cuda rand states and mallocs space for them, each state is thread safe
	*/
	hipSetDevice(0);
	hipEvent_t gpuStart, gpuEnd;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuEnd);
	printf("Size");
	for (int blocks = 1; blocks <= 128; blocks *= 2) {
		printf("\t%d", blocks);
	}
	printf("\n");
	for (long long tosses = 25000; tosses <= 10000000; tosses += 25000) {
		printf("%d", tosses);
		for (int blocks = 1; blocks <= 128; blocks *= 2) {
			long long N = tosses;
			int blockSize = 32;
			int numBlocks = blocks;// (int)(N + blockSize - 1) / blockSize;

			hiprandState_t* states;
			hipMalloc((void**)&states, blockSize * numBlocks * sizeof(hiprandState_t));

			long long total = 0;
			int* inCirc_d;
			int* inCirc = (int*)malloc(blockSize * numBlocks * sizeof(int));


			hipMalloc((void**)&inCirc_d, blockSize * numBlocks * sizeof(int));
			hipMemset(inCirc_d, 0, blockSize * numBlocks * sizeof(int));
			//CUDAErrorCheck();

			hipEventRecord(gpuStart);
			hipEventSynchronize(gpuStart);
			monteCarlo << <numBlocks, blockSize >> > (N, inCirc_d, states, time(NULL));
			hipEventRecord(gpuEnd);
			hipEventSynchronize(gpuEnd);
			hipDeviceSynchronize();

			float msUsed;
			hipEventElapsedTime(&msUsed, gpuStart, gpuEnd);
			//printf("%d	%lld	%f\n", blocks, tosses, msUsed);
			printf("\t%f", msUsed);


			//CUDAErrorCheck();

			hipMemcpy(inCirc, inCirc_d, blockSize * numBlocks * sizeof(int), hipMemcpyDeviceToHost);
			//CUDAErrorCheck();
			hipDeviceSynchronize();

			//sumArray(inCirc, N, &total, numBlocks, blockSize, onGpu);

			/*for (long long i = 0; i < blockSize * numBlocks; i++) {
				total += inCirc[i];
				//printf("i = %lld and count = %d\n", i, inCirc[i]);
			}*/


			//double pi_estimate = 4 * total / (double)N;

			//printf("Total in Circle = %lld\nEstimate of Pi = %lf\n", total, pi_estimate);

			hipFree(states);
			hipFree(inCirc_d);
		}
		printf("\n");
	}
	hipEventDestroy(gpuStart);
	hipEventDestroy(gpuEnd);

	hipDeviceReset();
	return 0;
}

